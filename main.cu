#include "hip/hip_runtime.h"
#pragma comment(lib, "cublas.lib")
#pragma comment(lib, "cudnn.lib")

#include <hip/hip_runtime.h>

#include "cuda/utils.hpp"
#include "cuda/memory.hpp"
#include "cuda/cublas.hpp"

#include "benchmark.hpp"

#include <iostream>
#include <chrono>
#include <random>
#include <vector>
#include <iomanip>

/* cpu implementations */
namespace cpu {
    template <class T>
    void vector_add(const T* first1, const T* first2, T* d_first, std::size_t N) {
        for (std::size_t i = 0; i < N; i++)
            d_first[i] = first1[i] + first2[i];
    }

    template <class T>
    void matrix_add(const T* first, const T* second, T* result, std::size_t nx, std::size_t ny) {
        for (std::size_t i = 0; i < ny; i++) {
            for (std::size_t j = 0; j < nx; j++) {
                const auto idx = i * nx + j;
                result[idx] = first[idx] + second[idx];
            }
        }
    }

    template <class T>
    void matrix_multiply(const T* first, const T* second, T* result, std::size_t n) {
        for (std::size_t i = 0; i < n; i++) {
            for (std::size_t j = 0; j < n; j++) {
                const auto idx = j * n + i;
                result[idx] = T(0);
                for (std::size_t k = 0; k < n; k++) {
                    const auto first_idx = i * n + k,
                               second_idx = k * n + j;
                    result[idx] += first[first_idx] * second[second_idx];
                }
            }
        }
    }
}

/* custom gpu implementations */
namespace gpu {
    template <class T>
    __global__ void vector_add(const T* first1, const T* first2, T* d_first, std::size_t N) {
        for(auto i : cuda::grid_stride_range(0, N))
            d_first[i] = first1[i] + first2[i];
    }

    template <class T>
    __global__ void matrix_add(const T* first, const T* second, T* result, std::size_t nx, std::size_t ny) {
        for (auto idx : cuda::grid_stride_range(0, nx * ny)) {
            result[idx] = first[idx] + second[idx];
        }
    }

    template <class T>
    __global__ void matrix_multiply(const T* first, const T* second, T* result, std::size_t n) {
        for (auto i : cuda::grid_stride_range_x(n)) {
            for (auto j : cuda::grid_stride_range_y(n)) {
                const auto idx = j * n + i;
                result[idx] = T(0); /* TODO CHECK PTX becaz using temporary variable slows down */
                for (std::size_t k = 0; k < n; k++) {
                    const auto first_idx = i * n + k,
                               second_idx = k * n + j;
                    result[idx] += first[first_idx] * second[second_idx];
                }
            }
        }        
    }
}

/* cublas implementation */
namespace cublas {
    template <class T>
    void matrix_multiply(cuda::cublas_handle& handle,  const T* first, const T* second, T* result, std::size_t n) {
        static_assert(std::is_same<T, float>::value, "uses hipblasSgemm; hence, requires T to be float");

        int in = static_cast<int>(n);
        const float alpha = 1.0, beta = 0.0;
        hipblasSgemm(handle.get(), HIPBLAS_OP_T, HIPBLAS_OP_T,
                    in, in, in,
                    &alpha,
                    first, in,
                    second, in,
                    &beta,
                    result, in);
        hipblasSgeam(handle.get(), HIPBLAS_OP_T, HIPBLAS_OP_N, in, in, &alpha, result, in, &beta, nullptr, in, result, in);        
    }

    template <class T>
    void matrix_add(cuda::cublas_handle& handle, const T* first, const T* second, T* result, std::size_t nx, std::size_t ny) {
        static_assert(std::is_same<T, float>::value, "uses hipblasSgeam; hence, requires T to be float");

        int inx = static_cast<int>(nx), iny = static_cast<int>(ny);
        const float alpha = 1.0, beta = 1.0;
        hipblasSgeam(handle.get(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                    iny, inx,
                    &alpha,
                    first, inx,
                    &beta,
                    second, inx,
                    result, inx);
    }
}

template <class T>
auto to_milliseconds(const T& duration) {
    return std::chrono::duration_cast<std::chrono::milliseconds>(duration);
}

template <class T, class ForwardItr>
auto check_result(ForwardItr first1, ForwardItr last1, ForwardItr first2, T ratio) {
    return std::mismatch(first1, last1, first2, [ratio](auto lhs, auto rhs) {
        return std::fabs(rhs - lhs) / std::min(rhs, lhs) < ratio;
    });
}

template <class Container>
void random_fill(Container& cont) {
    static std::random_device rd;
    static std::mt19937 rng(rd());
    static std::uniform_real_distribution<typename Container::value_type> dist(1.0, 1000.0);
    std::generate(std::begin(cont), std::end(cont), []() { return dist(rng); });
}

void test_matrix_multiply() {
    using T = float;

    constexpr std::size_t n = 1 << 12, size = n * n;

    /* generate sample data */
    std::vector<T> lhs(size), rhs(size);
    random_fill(lhs);
    random_fill(rhs);

    /* run on cpu */
    std::vector<T> cpu_result(size);
    auto cpu_time = benchmark([&lhs, &rhs, &cpu_result] () {
        //cpu::matrix_multiply(lhs.data(), rhs.data(), &cpu_result[0], n);
    });    
    std::cout << "CPU Time: " << to_milliseconds(cpu_time).count() << "ms" << std::endl;

    /* setup GPU */
    cuda::managed_ptr<T> d_lhs, d_rhs, d_result;
    auto gpu_prep_time = benchmark([&d_lhs, &d_rhs, &d_result, &lhs, &rhs]() {
        d_lhs.reset(size);
        d_rhs.reset(size);
        d_result.reset(size);
        cuda::memcpy(d_lhs, lhs.data(), size * sizeof(T));
        cuda::memcpy(d_rhs, rhs.data(), size * sizeof(T));
    });
    std::cout << "GPU Preparation Time: " << to_milliseconds(gpu_prep_time).count() << "ms" << std::endl;

    /* run custom GPU kernel */
    auto gpu_time = benchmark([&d_lhs, &d_rhs, &d_result]() {
        dim3 block(32, 32);
        dim3 grid((n + block.x - 1)/block.x, (n + block.y - 1)/block.y);
        cuda::launch_kernel(gpu::matrix_multiply<T>, grid, block, d_lhs.get(), d_rhs.get(), d_result.get(), n);
        /* cuda::launch_kernel(gpu::matrix_multiply<T>, d_lhs.get(), d_rhs.get(), d_result.get(), n); */
        cuda::device_synchronize();
    });

    std::vector<T> gpu_result(size);
    cuda::memcpy(&gpu_result[0], d_result, size * sizeof(T));
    std::cout << "GPU Time: " << to_milliseconds(gpu_time).count() << "ms" << std::endl;    
    
    auto pr = check_result(std::begin(cpu_result), std::end(cpu_result), std::begin(gpu_result), 0.001);
    bool match = (pr.first == std::end(cpu_result));
    std::cout << "CPU and GPU output " << (match ? "match" : "do not match") << std::endl;
    if (!match) {
        std::cout << std::setprecision(std::numeric_limits<T>::digits10 + 1);
        std::cout << "Mismatch: "<< *pr.first << " " << *pr.second << std::endl;
    }

    cuda::memset(d_result, 0, size * sizeof(T));
    cuda::device_synchronize();

    cuda::cublas_handle handle; /* declared outside because lazy initialization screws with the benchmarks */
    auto cublas_time = benchmark([&handle, &d_lhs, &d_rhs, &d_result]() {
        cublas::matrix_multiply(handle, d_lhs.get(), d_rhs.get(), d_result.get(), n);
        cuda::device_synchronize();
    });
    cuda::memcpy(&gpu_result[0], d_result, size * sizeof(T));
    std::cout << "CUBLAS Time: " << to_milliseconds(cublas_time).count() << "ms" << std::endl;    
    
    pr = check_result(std::begin(cpu_result), std::end(cpu_result), std::begin(gpu_result), T(0.02));
    match = (pr.first == std::end(cpu_result));
    std::cout << "CPU and CUBLAS output " << (match ? "match" : "do not match") << std::endl;
    if (!match) {
        std::cout << std::setprecision(std::numeric_limits<T>::digits10 + 1);
        std::cout << "Mismatch: "<< *pr.first << " " << *pr.second << std::endl;
    }  
}

void test_matrix_add() {
    using T = float;

    constexpr int n = 1 << 14, size = n * n;

    std::vector<T> lhs(size), rhs(size);
    random_fill(lhs);
    random_fill(rhs);

    std::vector<T> cpu_result(size);
    auto cpu_time = benchmark([&lhs, &rhs, &cpu_result] () {
        cpu::matrix_add(lhs.data(), rhs.data(), &cpu_result[0], n, n);
    });    
    std::cout << "CPU Time: " << to_milliseconds(cpu_time).count() << "ms" << std::endl;

    cuda::managed_ptr<T> d_lhs, d_rhs, d_result;
    auto gpu_prep_time = benchmark([&d_lhs, &d_rhs, &d_result, &lhs, &rhs]() {
        d_lhs.reset(size);
        d_rhs.reset(size);
        d_result.reset(size);
        cuda::memcpy(d_lhs, lhs.data(), size * sizeof(T));
        cuda::memcpy(d_rhs, rhs.data(), size * sizeof(T));
    });
    std::cout << "GPU Preparation Time: " << to_milliseconds(gpu_prep_time).count() << "ms" << std::endl;

    auto gpu_time = benchmark([&d_lhs, &d_rhs, &d_result]() {
        cuda::launch_kernel(gpu::matrix_add<T>, d_lhs.get(), d_rhs.get(), d_result.get(), n, n);
        cuda::device_synchronize();
    });
    std::vector<T> gpu_result(size);
    cuda::memcpy(&gpu_result[0], d_result, size * sizeof(T));
    std::cout << "GPU Time: " << to_milliseconds(gpu_time).count() << "ms" << std::endl;    
    
    auto pr = check_result(std::begin(cpu_result), std::end(cpu_result), std::begin(gpu_result), T(0.02));
    bool match = pr.first == std::end(cpu_result);
    std::cout << "CPU and GPU output " << (match ? "match" : "do not match") << std::endl;
    if (!match) {
        std::cout << std::setprecision(std::numeric_limits<T>::digits10 + 1);
        std::cout << "Mismatch: "<< *pr.first << " " << *pr.second << std::endl;
    }

    cuda::memset(d_result, 0, size * sizeof(T));
    cuda::device_synchronize();

    cuda::cublas_handle handle; /* declared outside because lazy initialization screws with the benchmarks */
    auto cublas_time = benchmark([&handle, &d_lhs,&d_rhs, &d_result]() {
        cublas::matrix_add(handle, d_lhs.get(), d_rhs.get(), d_result.get(), n, n);
        cuda::device_synchronize();
    });
    cuda::memcpy(&gpu_result[0], d_result, size * sizeof(T));
    std::cout << "CUBLAS Time: " << to_milliseconds(cublas_time).count() << "ms" << std::endl;    
    
    pr = check_result(std::begin(cpu_result), std::end(cpu_result), std::begin(gpu_result), T(0.02));
    match = (pr.first == std::end(cpu_result));
    std::cout << "CPU and CUBLAS output " << (match ? "match" : "do not match") << std::endl;
    if (!match) {
        std::cout << std::setprecision(std::numeric_limits<T>::digits10 + 1);
        std::cout << "Mismatch: "<< *pr.first << " " << *pr.second << std::endl;
    }
}

void test_vector_add() {
    using T = float;

    constexpr int N = 1 << 26;

    std::vector<T> lhs(N), rhs(N);
    random_fill(lhs);
    random_fill(rhs);

    std::vector<T> cpu_result(N);
    auto cpu_time = benchmark([&lhs, &rhs, &cpu_result] () {
        cpu::vector_add(lhs.data(), rhs.data(), &cpu_result[0], N);
    });    
    std::cout << "CPU Time: " << to_milliseconds(cpu_time).count() << "ms" << std::endl;

    cuda::managed_ptr<T> d_lhs, d_rhs, d_result;
    auto gpu_prep_time = benchmark([&d_lhs, &d_rhs, &d_result, &lhs, &rhs]() {
        d_lhs.reset(N);
        d_rhs.reset(N);
        d_result.reset(N);
        cuda::memcpy(d_lhs, lhs.data(), N * sizeof(T));
        cuda::memcpy(d_rhs, rhs.data(), N * sizeof(T));
    });
    std::cout << "GPU Preparation Time: " << to_milliseconds(gpu_prep_time).count() << "ms" << std::endl;

    auto gpu_time = benchmark([&d_lhs,&d_rhs, &d_result]() {
        cuda::launch_kernel(gpu::vector_add<T>, d_lhs.get(), d_rhs.get(), d_result.get(), N);
        cuda::device_synchronize();
    });
    std::vector<T> gpu_result(N);
    cuda::memcpy(&gpu_result[0], d_result, N * sizeof(T));
    std::cout << "GPU Time: " << to_milliseconds(gpu_time).count() << "ms" << std::endl;    
    
    auto pr = check_result(std::begin(cpu_result), std::end(cpu_result), std::begin(gpu_result), T(0.02));
    bool match = (pr.first == std::end(cpu_result));
    std::cout << "CPU and GPU output " << (match ? "match" : "do not match") << std::endl;
    if (!match) {
        std::cout << std::setprecision(std::numeric_limits<T>::digits10 + 1);
        std::cout << "Mismatch: "<< *pr.first << " " << *pr.second << std::endl;
    }
}

int main() {
    int dev = 0;
    hipDeviceProp_t properties;
    CHECK_CUDA(hipGetDeviceProperties(&properties, dev));
    std::cout << "Device: " << dev << ", " << properties.name << '\n' << std::endl;

    CHECK_CUDA(hipSetDevice(dev));
    CHECK_CUDA(hipFree(0)); /* establish context beforehand so that the benchmarks are not disturbed */

    std::cout << "Vector Addition:\n";
    test_vector_add();
    std::cout << std::endl;

    std::cout << "Matrix Addition:\n";
    test_matrix_add();
    std::cout << std::endl;

    std::cout << "Matrix Multiplication:\n";
    test_matrix_multiply();

    CHECK_CUDA(hipDeviceReset());
    return 0;
}